#include "hip/hip_runtime.h"
#include "reference_string.cu"
#include <assert.h> 
#include <iostream>
#include <algorithm>
#include <memory>
#include <fstream>
#include <string>

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg;

// Kernel launch parameters
static constexpr size_t BLOCKS = 1;
static constexpr size_t THREADS = 4;
static constexpr size_t POINTS = 1 << 10;

/* -------------------------- Kernel Functions For Finite Field Tests ---------------------------------------------- */

// Initialize points and scalars
__global__ void initialize_simple_double_and_add_field(
uint64_t *a, uint64_t *b, uint64_t *expect_x) {
    fq_gpu point{ 0x2523b6fa3956f038, 0x158aa08ecdd9ec1d, 0xf48216a4c74738d4, 0x2514cc93d6f0a1bf };
    fq_gpu scalar{ 0x09, 0x0, 0x0, 0x0 };
    fq_gpu expected_x{ 0xe57e2642f120824e, 0x34d7259cc9fe8db6, 0x46b12983f878ed43, 0x2b615a81474beec5 };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = scalar.data[i];
        b[i] = point.data[i];
        expect_x[i] = expected_x.data[i];
    }
}

// Simple montgomery multiplication as a baseline reference
__global__ void simple_multiplication_field(uint64_t *scalar, uint64_t *point, uint64_t *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::to_monty(scalar[tid], scalar[tid]);  
        fq_gpu::mul(point[tid], scalar[tid], result[tid]); 
    }
}

// Native approach for computing scalar mutliplication with time complexity: O(2^k)
// nP = P + P ... + P 
__global__ void naive_double_and_add_field(uint64_t *scalar, uint64_t *point, uint64_t *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::add(point[tid], point[tid], result[tid]);     
        fq_gpu::add(point[tid], result[tid], result[tid]);
        fq_gpu::add(point[tid], result[tid], result[tid]);
        fq_gpu::add(point[tid], result[tid], result[tid]);
        fq_gpu::add(point[tid], result[tid], result[tid]);
        fq_gpu::add(point[tid], result[tid], result[tid]);
        fq_gpu::add(point[tid], result[tid], result[tid]);
        fq_gpu::add(point[tid], result[tid], result[tid]);
    }
}

// Double and add implementation using bit-decomposition with time complexity: O(2^k / 2)
__global__ void double_and_add_half_field(uint64_t *scalar, uint64_t *point, uint64_t *result) {
    fq_gpu R;
    fq_gpu Q;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        // Initialize 'R' to the identity element, Q to the curve point
        fq_gpu::load(0, R.data[tid]); 
        fq_gpu::load(point[tid], Q.data[tid]);

        // Loop unrolling
        fq_gpu::add(R.data[tid], Q.data[tid], R.data[tid]);   // P
        fq_gpu::add(R.data[tid], R.data[tid], R.data[tid]);   // 2P
        fq_gpu::add(R.data[tid], R.data[tid], R.data[tid]);   // 4P
        fq_gpu::add(R.data[tid], R.data[tid], R.data[tid]);   // 8P 
        fq_gpu::add(R.data[tid], Q.data[tid], R.data[tid]);   // 9P 
    }
    
    // Store the final value of R into the result array for this limb
    fq_gpu::load(R.data[tid], result[tid]);
}

// Double and add implementation using bit-decomposition with time complexity: O(k)
__global__ void double_and_add_field(uint64_t *scalar, uint64_t *point, uint64_t *result) {
    fq_gpu R;
    fq_gpu Q;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        // Initialize 'R' to the identity element, Q to the curve point
        fq_gpu::load(0, R.data[tid]); 
        fq_gpu::load(point[tid], Q.data[tid]);
        
        for (int i = 3; i >= 0; i--) {
            // Performs bit-decompositon by traversing the bits of the scalar from MSB to LSB
            // and extracting the i-th bit of scalar in limb.
            if (((scalar[0] >> i) & 1) ? 1 : 0)
                fq_gpu::add(R.data[tid], Q.data[tid], R.data[tid]);  
            if (i != 0) 
                fq_gpu::add(R.data[tid], R.data[tid], R.data[tid]); 
        }
    }
    
    // Store the final value of R into the result array for this limb
    fq_gpu::load(R.data[tid], result[tid]);
}

/* -------------------------- Kernel Functions For Elliptic Curve Tests ---------------------------------------------- */

// Initialize points and scalars
__global__ void initialize_simple_double_and_add_curve(
uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *d, uint64_t *expect_x, uint64_t *expect_y, uint64_t *expect_z) {
    fq_gpu point_x{ 0x184b38afc6e2e09a, 0x4965cd1c3687f635, 0x334da8e7539e71c4, 0xf708d16cfe6e14 };
    fq_gpu point_y{ 0x2a6ff6ffc739b3b6, 0x70761d618b513b9, 0xbf1645401de26ba1, 0x114a1616c164b980 };
    fq_gpu point_z{ 0x10143ade26bbd57a, 0x98cf4e1f6c214053, 0x6bfdc534f6b00006, 0x1875e5068ababf2c };
    fq_gpu scalar{ 0x09, 0x0, 0x0, 0x0 };
    fq_gpu expected_x{ 0xb95b0df1fafbbf24, 0x848b28a0183c5cb7, 0x8158042f18cfd297, 0x124a5cddf43c0bc2 };
    fq_gpu expected_y{ 0x5769f9d04cd40953, 0x15f951f775281d56, 0x8b6b9be09b2bcd61, 0x1d2dbd94949735db };
    fq_gpu expected_z{ 0x5644e29729c3e1ce, 0xbf97116e02fc9117, 0x2f18c34822c7b2cd, 0x867c7e32dc19f38 };

    for (int i = 0; i < LIMBS; i++) {
        a[i] = point_x.data[i];
        b[i] = point_y.data[i];
        c[i] = point_z.data[i];
        d[i] = scalar.data[i];
        expect_x[i] = expected_x.data[i];
        expect_y[i] = expected_y.data[i];
        expect_z[i] = expected_z.data[i];
    }
}

// Native approach for computing scalar mutliplication with time complexity: O(2^k)
// nP = P + P ... + P 
__global__ void naive_double_and_add_curve(uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *d, var *res_x, var *res_y, var *res_z) {
    g1::element ec;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::to_monty(a[tid], ec.x.data[tid]);
        fq_gpu::to_monty(b[tid], ec.y.data[tid]);
        fq_gpu::to_monty(c[tid], ec.z.data[tid]);

        // Jacobian addition
        g1::add(
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // ec + ec = 2ec
        if (fq_gpu::is_zero(res_x[tid]) && fq_gpu::is_zero(res_y[tid]) && fq_gpu::is_zero(res_z[tid])) {
            g1::doubling(
                ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
                res_x[tid], res_y[tid], res_z[tid]
            );
        }

        // 2ec + ec = 3ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 3ec + ec = 4ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 4ec + ec = 5ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 5ec + ec = 6ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 6ec + ec = 7ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 7ec + ec = 8ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 8ec + ec = 9ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);
    }
}

// Double and add implementation using bit-decomposition with time complexity: O(2^k / 2)
__global__ void double_and_add_half_curve(uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *d, var *res_x, var *res_y, var *res_z) {
    g1::element ec;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::to_monty(a[tid], ec.x.data[tid]);
        fq_gpu::to_monty(b[tid], ec.y.data[tid]);
        fq_gpu::to_monty(c[tid], ec.z.data[tid]);

        // Jacobian addition
        g1::add(
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // ec + ec = 2ec
        if (fq_gpu::is_zero(res_x[tid]) && fq_gpu::is_zero(res_y[tid]) && fq_gpu::is_zero(res_z[tid])) {
            g1::doubling(
                ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
                res_x[tid], res_y[tid], res_z[tid]
            );
        }

        // 2ec + 2ec = 4ec
        g1::doubling(
            res_x[tid], res_y[tid], res_z[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 4ec + 4ec = 8ec
        g1::doubling(
            res_x[tid], res_y[tid], res_z[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        // 8ec + ec = 9ec
        g1::add(
            res_x[tid], res_y[tid], res_z[tid], 
            ec.x.data[tid], ec.y.data[tid], ec.z.data[tid], 
            res_x[tid], res_y[tid], res_z[tid]
        );

        fq_gpu::from_monty(res_x[tid], res_x[tid]);
        fq_gpu::from_monty(res_y[tid], res_y[tid]);
        fq_gpu::from_monty(res_z[tid], res_z[tid]);
    }
}

// Double and add implementation using bit-decomposition with time complexity: O(k)
__global__ void double_and_add_curve(
uint64_t *point_x, uint64_t *point_y, uint64_t *point_z, uint64_t *scalar, var *res_x, var *res_y, var *res_z) {
    g1::element R;
    g1::element Q;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < LIMBS) {
        // Initialize 'R' to the identity element, Q to the curve point
        fq_gpu::load(0, R.x.data[tid]); 
        fq_gpu::load(0, R.y.data[tid]); 
        fq_gpu::load(0, R.z.data[tid]); 

        fq_gpu::load(point_x[tid], Q.x.data[tid]);
        fq_gpu::load(point_y[tid], Q.y.data[tid]);
        fq_gpu::load(point_z[tid], Q.z.data[tid]);

        fq_gpu::to_monty(point_x[tid], Q.x.data[tid]);
        fq_gpu::to_monty(point_y[tid], Q.y.data[tid]);
        fq_gpu::to_monty(point_z[tid], Q.z.data[tid]);

        for (int i = 3; i >= 0; i--) {
            // Performs bit-decompositon by traversing the bits of the scalar from MSB to LSB
            // and extracting the i-th bit of scalar in limb.
            if (((scalar[0] >> i) & 1) ? 1 : 0)
                g1::add(
                    R.x.data[tid], R.y.data[tid], R.z.data[tid], 
                    Q.x.data[tid], Q.y.data[tid], Q.z.data[tid], 
                    R.x.data[tid], R.y.data[tid], R.z.data[tid]
                );
            if (i != 0) 
                g1::doubling(
                    R.x.data[tid], R.y.data[tid], R.z.data[tid], 
                    R.x.data[tid], R.y.data[tid], R.z.data[tid]
                );
        }
    }
    
    // Store the final value of R into the result array for this limb
    fq_gpu::load(R.x.data[tid], res_x[tid]);
    fq_gpu::load(R.y.data[tid], res_y[tid]);
    fq_gpu::load(R.z.data[tid], res_z[tid]);

    fq_gpu::from_monty(res_x[tid], res_x[tid]);
    fq_gpu::from_monty(res_y[tid], res_y[tid]);
    fq_gpu::from_monty(res_z[tid], res_z[tid]);
}

/* -------------------------- Kernel Functions For Vector of Finite Field Tests ---------------------------------------------- */

// Convert result from montgomery form
__global__ void convert(fq_gpu *point, uint64_t *result) {
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        fq_gpu::from_monty(point[0].data[tid], result[tid]);
    }

}

// Naive double and add using sequential implementation
__global__ void naive_double_and_add_field_vector_simple(fq_gpu *point, fq_gpu *result_vec, uint64_t *result) { 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    fq_gpu res{ 0, 0, 0, 0 };
    for (int i = 0; i < 1024; i++) {
        fq_gpu::add(res.data[tid], point[i].data[tid], res.data[tid]);
    }
    fq_gpu::load(res.data[tid], result[tid]);
    fq_gpu::from_monty(result[tid], result[tid]);
}

// Naive double and add using multiple kernel invocations with block-level grandularity
__global__ void naive_double_and_add_field_vector(fq_gpu *point, fq_gpu *result_vec, uint64_t *result) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    fq_gpu::add(
        point[blockIdx.x * 2].data[threadIdx.x], point[(blockIdx.x * 2) + 1].data[threadIdx.x], result_vec[blockIdx.x].data[threadIdx.x]
    );
    
    if (threadIdx.x == 0) {
        fq_gpu::load(point[0].data[0], result[0]);
        fq_gpu::load(point[0].data[1], result[1]);
        fq_gpu::load(point[0].data[2], result[2]);
        fq_gpu::load(point[0].data[3], result[3]);
    }
}

/* -------------------------- Kernel Functions For Vector of Elliptic Curve Tests ---------------------------------------------- */


/* -------------------------- Helper Functions ---------------------------------------------- */

// Read curve points
template <class B>
B* read_points() {
    fq_gpu *points = new fq_gpu[POINTS];
    std::ifstream myfile ("../src/aztec/gpu/benchmark/tests/msm/points.txt"); 

    int i = 0;
    if ( myfile.is_open() ) {     
        for (size_t i = 0; i < POINTS * 4; ++i) {
            for (size_t j = 0; j < 4; j++) {
                myfile >> points[i].data[j];
            }
        }
    }
    return points;
} 

// Read scalars
template <class B>
B* read_scalars() {
    uint64_t temp[POINTS * 4];
    fr_gpu *scalars = new fr_gpu[POINTS];
    uint64_t scalar;

    // File stream
    ifstream stream;
    stream.open("../src/aztec/gpu/msm/scalars/scalars.txt", ios::in);

    // Read scalars
    if (stream.is_open()) {   
        int i = 0;  
        while (stream.good()) { 
            stream >> scalar;
            temp[i] = scalar;
            i++;
        }
    }
    
    for (size_t i = 0; i < (sizeof(temp) / sizeof(uint64_t)) / 4; ++i) {    
        fr_gpu element{ temp[i], temp[i + 1], temp[i + 2], temp[i + 3] };
        scalars[i] = element;        
    }
        
    return scalars;
}

// Print results
void print_field_tests(var *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print results for each limb
    printf("result[0] is: %zu\n", result[0]);
    printf("result[1] is: %zu\n", result[1]);
    printf("result[2] is: %zu\n", result[2]);
    printf("result[3] is: %zu\n\n", result[3]);
}

void print_curve_tests(var *res_x, var *res_y, var *res_z) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print results for each limb
    printf("res_x[0] is: %zu\n", res_x[0]);
    printf("res_x[1] is: %zu\n", res_x[1]);
    printf("res_x[2] is: %zu\n", res_x[2]);
    printf("res_x[3] is: %zu\n\n", res_x[3]);

    printf("res_y[0] is: %zu\n", res_y[0]);
    printf("res_y[1] is: %zu\n", res_y[1]);
    printf("res_y[2] is: %zu\n", res_y[2]);
    printf("res_y[3] is: %zu\n\n", res_y[3]);

    printf("res_z[0] is: %zu\n", res_z[0]);
    printf("res_z[1] is: %zu\n", res_z[1]);
    printf("res_z[2] is: %zu\n", res_z[2]);
    printf("res_z[3] is: %zu\n\n", res_z[3]);
}

void print_field_vector_tests(fq_gpu *result_vec) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print results for each limb
    printf("result[0] is: %zu\n", result_vec[0].data[0]);
    printf("result[1] is: %zu\n", result_vec[0].data[1]);
    printf("result[2] is: %zu\n", result_vec[0].data[2]);
    printf("result[3] is: %zu\n\n", result_vec[0].data[3]);
}

// Assert statements
void assert_checks(var *expected, var *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Assert clause
    assert(expected[0] == result[0]);
    assert(expected[1] == result[1]);
    assert(expected[2] == result[2]);
    assert(expected[3] == result[3]);
}

/* -------------------------- Executing Initialization and Workload Kernels ---------------------------------------------- */

// Execute kernel with finite field elements
void execute_kernels_finite_fields(
var *a, var *b, var *c, var *d, var *result, var *res_x, var *res_y, var *res_z, var *expect_x,  var *expect_y,  var *expect_z) {    
    initialize_simple_double_and_add_field<<<BLOCKS, THREADS>>>(a, b, expect_x);
    
    double_and_add_field<<<BLOCKS, THREADS>>>(a, b, result);
    assert_checks(expect_x, result);
    print_field_tests(result);

    simple_multiplication_field<<<BLOCKS, THREADS>>>(a, b, result);
    assert_checks(expect_x, result);
    print_field_tests(result);

    naive_double_and_add_field<<<BLOCKS, THREADS>>>(a, b, result);
    assert_checks(expect_x, result);
    print_field_tests(result);
    
    double_and_add_half_field<<<BLOCKS, THREADS>>>(a, b, result);
    assert_checks(expect_x, result);
    print_field_tests(result);
}

/**
 * Execute kernel with curve elements
 */
void execute_kernels_curve(
var *a, var *b, var *c, var *d, var *result, var *res_x, var *res_y, var *res_z, var *expect_x, var *expect_y, var *expect_z) {    
    initialize_simple_double_and_add_curve<<<BLOCKS, THREADS>>>(a, b, c, d, expect_x, expect_y, expect_z);

    naive_double_and_add_curve<<<BLOCKS, THREADS>>>(a, b, c, d, res_x, res_y, res_z);
    print_curve_tests(res_x, res_y, res_z);

    double_and_add_half_curve<<<BLOCKS, THREADS>>>(a, b, c, d, res_x, res_y, res_z);
    assert_checks(expect_x, res_x);
    assert_checks(expect_y, res_y);
    assert_checks(expect_z, res_z);
    print_curve_tests(res_x, res_y, res_z);

    double_and_add_curve<<<BLOCKS, THREADS>>>(a, b, c, d, res_x, res_y, res_z);
    assert_checks(expect_x, res_x);
    assert_checks(expect_y, res_y);
    assert_checks(expect_z, res_z);
    print_curve_tests(res_x, res_y, res_z);
}

/**
 * Execute kernel with vector of finite field elements
 */
void execute_kernels_finite_fields_vector(
var *a, var *b, var *c, var *d, var *result, var *res_x, var *res_y, var *res_z, var *expect_x, var *expect_y, var *expect_z) {    
    // Read curve points and scalars
    fr_gpu *scalars = read_scalars<fr_gpu>();
    fq_gpu *points = read_points<fq_gpu>();

    // Define pointers to uint64_t type
    fq_gpu *points_alloc, *scalar_alloc, *result_vec;

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&points_alloc, LIMBS * POINTS * sizeof(var));
    hipMallocManaged(&scalar_alloc, LIMBS * POINTS * sizeof(var));
    hipMallocManaged(&result_vec, LIMBS * POINTS * sizeof(var));

    // Load points
    for (int i = 0; i < POINTS; i++) {
        for (int j = 0; j < LIMBS; j++) {
            points_alloc[i].data[j] = points[i].data[j];
        }
    }

    // Load expected result
    expect_x[0] = 0x2ABC1016AF87ED0;
    expect_x[1] = 0xB606DF3AF98259F3;
    expect_x[2] = 0x9EE7391E20B296B4;
    expect_x[3] = 0x21E559B660EDBD92;

    naive_double_and_add_field_vector_simple<<<1, 4>>>(points_alloc, result_vec, result);
    assert_checks(expect_x, result);
    print_field_tests(result);

    // the issue with this is it requires log(NUM_POINTS) kernel invocations
    naive_double_and_add_field_vector<<<1024, 4>>>(points_alloc, result_vec, result);
    naive_double_and_add_field_vector<<<512, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<256, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<128, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<64, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<32, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<16, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<8, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<4, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<2, 4>>>(result_vec, result_vec, result);
    naive_double_and_add_field_vector<<<1, 4>>>(result_vec, result_vec, result);
    convert<<<1, 4>>>(result_vec, result);
    print_field_tests(result);
}

/* -------------------------- Main Entry Function ---------------------------------------------- */

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define pointers to uint64_t type
    var *a, *b, *c, *d, *result, *res_x, *res_y, *res_z, *expect_x, *expect_y, *expect_z;

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS * sizeof(var));
    hipMallocManaged(&b, LIMBS * sizeof(var));
    hipMallocManaged(&c, LIMBS * sizeof(var));
    hipMallocManaged(&d, LIMBS * sizeof(var));
    hipMallocManaged(&result, LIMBS * sizeof(var));
    hipMallocManaged(&res_x, LIMBS * sizeof(var));
    hipMallocManaged(&res_y, LIMBS * sizeof(var));
    hipMallocManaged(&res_z, LIMBS * sizeof(var));
    hipMallocManaged(&expect_x, LIMBS * sizeof(var));
    hipMallocManaged(&expect_y, LIMBS * sizeof(var));
    hipMallocManaged(&expect_z, LIMBS * sizeof(var));

    // Execute kernel functions
    // execute_kernels_finite_fields(a, b, c, d, result, res_x, res_y, res_z, expect_x, expect_y, expect_z);
    // execute_kernels_curve(a, b, c, d, result, res_x, res_y, res_z, expect_x, expect_y, expect_z);
    execute_kernels_finite_fields_vector(a, b, c, d, result, res_x, res_y, res_z, expect_x, expect_y, expect_z);

    // Successfull execution of unit tests
    cout << "******* All 'MSM' unit tests passed! **********" << endl;

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(result);

    cout << "Completed sucessfully!" << endl;

    return 0;
}