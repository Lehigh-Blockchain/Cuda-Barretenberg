#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include "common.cuh"

namespace pippenger_common {

/**
 * Kernel function for "Pippenger's Bucket Method"
 */
__global__ void pippenger(
const point_t* points, size_t npoints, const scalar_t *scalars_, 
bucket_t(* buckets)[NWINS][1<<WBITS], bucket_t(* ret)[NWINS][NTHREADS][2]) {

}

}