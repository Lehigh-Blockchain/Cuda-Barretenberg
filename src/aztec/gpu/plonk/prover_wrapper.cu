#include "hip/hip_runtime.h"
#include "prover_wrapper.cuh"

/**
 * Construct proof by running prover rounds
 *
 * @return proof
 * */
waffle::plonk_proof &prover_wrapper::ProverWrapper::construct_proof() {
    cout << "Entered virtual construct_proof()" << endl;
    
    // Execute init round. Randomize witness polynomials
    execute_preamble_round();
    queue.process_queue();

    // Compute wire precommitments and sometimes random widget round commitments
    execute_first_round();
    queue.process_queue();

    // Fiat-Shamir eta + execute random widgets
    execute_second_round();
    queue.process_queue();

    // Fiat-Shamir beta, execute random widgets (Permutation 
    // widget is executed here) and fft the witnesses
    execute_third_round();
    queue.process_queue();

    execute_fourth_round();
    queue.process_queue();

    execute_fifth_round();
    execute_sixth_round();
    queue.process_queue();

    return export_proof();
}