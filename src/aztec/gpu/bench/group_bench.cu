#include "hip/hip_runtime.h"
#include "../fields/group.cu"

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg;

static constexpr size_t LIMBS_NUM = 4;
static constexpr size_t BLOCKS = 1;
static constexpr size_t THREADS = 1;

/* -------------------------- Mixed Addition ---------------------------------------------- */

__global__ void initialize_mixed_add_check_against_constants(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res) {
    fq_gpu a_x{ 0x92716caa6cac6d26, 0x1e6e234136736544, 0x1bb04588cde00af0, 0x9a2ac922d97e6f5 };
    fq_gpu a_y{ 0x9e693aeb52d79d2d, 0xf0c1895a61e5e975, 0x18cd7f5310ced70f, 0xac67920a22939ad };
    fq_gpu a_z{ 0xfef593c9ce1df132, 0xe0486f801303c27d, 0x9bbd01ab881dc08e, 0x2a589badf38ec0f9 };
    fq_gpu b_x{ 0xa1ec5d1398660db8, 0x6be3e1f6fd5d8ab1, 0x69173397dd272e11, 0x12575bbfe1198886 };
    fq_gpu b_y{ 0xcfbfd4441138823e, 0xb5f817e28a1ef904, 0xefb7c5629dcc1c42, 0x1a9ed3d6f846230e };
    fq_gpu exp_x{ 0x2a9d0201fccca20, 0x36f969b294f31776, 0xee5534422a6f646, 0x911dbc6b02310b6 };
    fq_gpu exp_y{ 0x14c30aaeb4f135ef, 0x9c27c128ea2017a1, 0xf9b7d80c8315eabf, 0x35e628df8add760 };
    fq_gpu exp_z{ 0xa43fe96673d10eb3, 0x88fbe6351753d410, 0x45c21cc9d99cb7d, 0x3018020aa6e9ede5 };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
        x[i] = b_x.data[i];
        y[i] = b_y.data[i];
    }
}

__global__ void mixed_add_check_against_constants(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res) {
    g1::element lhs;
    g1::affine_element rhs;
    g1::element result;
    g1::element expected;
    
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        lhs.x.data[tid] = fq_gpu::to_monty(a[tid], res[tid]);
        lhs.y.data[tid] = fq_gpu::to_monty(b[tid], res[tid]);
        lhs.z.data[tid] = fq_gpu::to_monty(c[tid], res[tid]);
        rhs.x.data[tid] = fq_gpu::to_monty(x[tid], res[tid]);
        rhs.y.data[tid] = fq_gpu::to_monty(y[tid], res[tid]);

        g1::mixed_add(lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], rhs.x.data[tid], rhs.y.data[tid], expected_x[tid], expected_y[tid], expected_z[tid]);
        
        expected_x[tid] = fq_gpu::from_monty(expected_x[tid], expected_x[tid]);
        expected_y[tid] = fq_gpu::from_monty(expected_y[tid], expected_y[tid]);
        expected_z[tid] = fq_gpu::from_monty(expected_z[tid], expected_z[tid]);

        // EXPECT_EQ(result == expected, true);
    }
}

/* -------------------------- Doubling ---------------------------------------------- */

__global__ void initialize_dbl_check_against_constants(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res) {
    fq_gpu a_x{ 0x8d1703aa518d827f, 0xd19cc40779f54f63, 0xabc11ce30d02728c, 0x10938940de3cbeec };
    fq_gpu a_y{ 0xcf1798994f1258b4, 0x36307a354ad90a25, 0xcd84adb348c63007, 0x6266b85241aff3f };
    fq_gpu a_z{ 0xe213e18fd2df7044, 0xb2f42355982c5bc8, 0xf65cf5150a3a9da1, 0xc43bde08b03aca2 };
    fq_gpu exp_x{ 0xd5c6473044b2e67c, 0x89b185ea20951f3a, 0x4ac597219cf47467, 0x2d00482f63b12c86 };
    fq_gpu exp_y{ 0x4e7e6c06a87e4314, 0x906a877a71735161, 0xaa7b9893cc370d39, 0x62f206bef795a05 };
    fq_gpu exp_z{ 0x8813bdca7b0b115a, 0x929104dffdfabd22, 0x3fff575136879112, 0x18a299c1f683bdca };

    for (int i = 0; i < LIMBS_NUM; i++) {
        a[i] = a_x.data[i];
        b[i] = a_y.data[i];
        c[i] = a_z.data[i];
    }
}

__global__ void dbl_check_against_constants(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res) {
    g1::element lhs;
    g1::element result;
    g1::element expected;
    
    // Calculate global thread ID, and boundry check
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (tid < LIMBS) {
        lhs.x.data[tid] = fq_gpu::to_monty(a[tid], res[tid]);
        lhs.y.data[tid] = fq_gpu::to_monty(b[tid], res[tid]);
        lhs.z.data[tid] = fq_gpu::to_monty(c[tid], res[tid]);

        g1::doubling(lhs.x.data[tid], lhs.y.data[tid], lhs.z.data[tid], expected_x[tid], expected_y[tid], expected_z[tid]);
        g1::doubling(expected_x[tid], expected_y[tid], expected_z[tid], expected_x[tid], expected_y[tid], expected_z[tid]);
        g1::doubling(expected_x[tid], expected_y[tid], expected_z[tid], expected_x[tid], expected_y[tid], expected_z[tid]);

        // Converting from monty form gets back the correct result
        expected_x[tid] = fq_gpu::from_monty(expected_x[tid], res[tid]);
        expected_y[tid] = fq_gpu::from_monty(expected_y[tid], res[tid]);
        expected_z[tid] = fq_gpu::from_monty(expected_z[tid], res[tid]);

        // EXPECT_EQ(result == expected, true);
    }
}

/* -------------------------- Main -- Executing Kernels ---------------------------------------------- */

void execute_kernels(var *a, var *b, var *c, var *x, var *y, var *z, var *expected_x, var *expected_y, var *expected_z, var *res) {
    // Initialization kernels
    // initialize_mixed_add_check_against_constants<<<BLOCKS, THREADS>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z, res);
    initialize_dbl_check_against_constants<<<BLOCKS, THREADS>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z, res);

    // Workload kernels
    // mixed_add_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z, res);
    dbl_check_against_constants<<<BLOCKS, LIMBS_NUM>>>(a, b, c, x, y, z, expected_x, expected_y, expected_z, res);
}

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define pointers to uint64_t type
    var *a, *b, *c, *x, *y, *z, *expected_x, *expected_y, *expected_z, *res;    

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS_NUM * sizeof(uint64_t));
    hipMallocManaged(&b, LIMBS_NUM * sizeof(uint64_t));
    hipMallocManaged(&c, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&x, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&y, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&z, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&expected_x, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&expected_y, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&expected_z, LIMBS * sizeof(uint64_t));
    hipMallocManaged(&res, LIMBS * sizeof(uint64_t));

    // Execute kernel functions
    execute_kernels(a, b, c, x, y, z, expected_x, expected_y, expected_z, res);

    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print results
    // printf("res[0] is: %zu\n", res[0]);
    // printf("res[1] is: %zu\n", res[1]);
    // printf("res[2] is: %zu\n", res[2]);
    // printf("res[3] is: %zu\n", res[3]);

    printf("expected_x[0] is: %zu\n", expected_x[0]);
    printf("expected_x[1] is: %zu\n", expected_x[1]);
    printf("expected_x[2] is: %zu\n", expected_x[2]);
    printf("expected_x[3] is: %zu\n", expected_x[3]);

    printf("expected_y[0] is: %zu\n", expected_y[0]);
    printf("expected_y[1] is: %zu\n", expected_y[1]);
    printf("expected_y[2] is: %zu\n", expected_y[2]);
    printf("expected_y[3] is: %zu\n", expected_y[3]);

    printf("expected_z[0] is: %zu\n", expected_z[0]);
    printf("expected_z[1] is: %zu\n", expected_z[1]);
    printf("expected_z[2] is: %zu\n", expected_z[2]);
    printf("expected_z[3] is: %zu\n", expected_z[3]);

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipFree(expected_x);
    hipFree(expected_y);
    hipFree(expected_z);
    hipFree(res);

    cout << "Completed sucessfully!" << endl;

    return 0;
}