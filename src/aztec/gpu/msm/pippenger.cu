#include "hip/hip_runtime.h"
#include "pippenger.cuh"
#include <iostream>
#include <memory>
#include <iostream>
#include <fstream>
#include <string>
#include <chrono>

using namespace std;

namespace pippenger_common {

/**
 * Entry point into "Pippenger's Bucket Method"
 */ 
template <class A, class S, class J>
Context<bucket_t, point_t, scalar_t, affine_t>* msm_t<A, S, J>::pippenger_initialize(A* points) {
    try {
        // Initialize context object 
        Context<bucket_t, point_t, scalar_t, affine_t> *context = new Context<bucket_t, point_t, scalar_t, affine_t>();

        // Initialize MSM parameters
        context->pipp = context->pipp.initialize_msm(context->pipp, NUM_POINTS);    

        // Allocate GPU storage for bases, scalars, and buckets 
        context->d_points_idx = context->pipp.allocate_bases(context->pipp);
        context->d_buckets_idx = context->pipp.allocate_buckets(context->pipp);

        for (size_t i = 0; i < NUM_BATCH_THREADS; i++) {
            context->d_scalar_idx[i] = context->pipp.allocate_scalars(context->pipp);
        }

        // Allocate pinned memory on host for scalars
        CUDA_WRAPPER(hipHostMalloc(&context->h_scalars, context->pipp.get_size_scalars(context->pipp)));

        // Transfer bases to device
        context->pipp.transfer_bases_to_device(context->pipp, context->d_points_idx, points);
    
        // Create results container
        context->result0 = context->pipp.result_container(context->pipp);
        context->result1 = context->pipp.result_container(context->pipp);

        // Return initialized context object
        return context;
    }
    catch (hipError_t) {
        cout << "Failed to initialize MSM." << endl;
        throw;
    }
}

/**
 * Perform MSM
 */ 
template <class A, class S, class J>
void msm_t<A, S, J>::pippenger_execute(Context<bucket_t, point_t, scalar_t, affine_t> *context, size_t num_points, A* points) {
    // Read scalars
    fr_gpu *scalars;
    hipMallocManaged(&scalars, NUM_POINTS * LIMBS * sizeof(uint64_t));
    context->pipp.read_scalars(scalars);
    
    // Create auxilary stream
    stream_t aux_stream(context->pipp.device);

    try {        
        // Store results
        typename pipp_t::result_container_t *kernel_result = &context->result0;
        typename pipp_t::result_container_t *accumulation_result = &context->result1;

        size_t d_scalars_xfer = context->d_scalar_idx[0];
        size_t d_scalars_compute = context->d_scalar_idx[1];

        // Create a channel_t object from thread pool
        channel_t<size_t> channel;

        size_t scalar_size = context->pipp.get_size_scalars(context->pipp);
        
        // Overlap bucket computation on the GPU with transfer of scalars
        int batch = 1;
        int work = 0;

        // Transfer scalars to device
        context->pipp.transfer_scalars_to_device(context->pipp, context->d_scalar_idx[1], scalars, aux_stream);

        // Synchronize cuda stream with CPU thread, blocking execution until stream completed all operations
        CUDA_WRAPPER(hipStreamSynchronize(aux_stream));

        // Launch kernel
        context->pipp.launch_kernel(context->pipp, context->d_points_idx, context->d_scalar_idx[1], context->d_buckets_idx);
    }
    catch (hipError_t) {
        cout << "Failed executing multi-scalar multiplication!" << endl;
        throw;
    }
}

/**
 * Perform naive MSM
 */ 
template <class A, class S, class J>
void msm_t<A, S, J>::naive_msm(Context<bucket_t,point_t,scalar_t,affine_t> *context, size_t npoints, A *points) {
    fr_gpu *d_scalars;
    J *j_points;
    J *final_result;
    J *result;

    // Allocate cuda memory 
    hipMallocManaged(&d_scalars, NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&j_points, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&result, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&final_result, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));

    // Read points and scalars
    context->pipp.read_jacobian_curve_points(j_points);
    context->pipp.read_scalars(d_scalars);

    // Naive MSM method
    msm_naive_kernel<<<8, 512>>>(j_points, d_scalars, result, NUM_POINTS);

    // Sum reduction 
    sum_reduction<<<1, 4>>>(result, final_result);

    // Final accumulation
    sum_reduction_accumulate<<<1, 4>>>(final_result, final_result);
    hipDeviceSynchronize();
    
    // Print results
    context->pipp.print_result(final_result);
}

/**
 * Perform MSM Bucket Method
 */ 
template <class A, class S, class J>
void msm_t<A, S, J>::msm_bucket_method(Context<bucket_t,point_t,scalar_t,affine_t> *context, size_t npoints, A *points) {
    J *j_points;
    S *d_scalars;
    J *result;

    // Allocate unified memory
    hipMallocManaged(&j_points, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&d_scalars, NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&result, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));

    // Calculate maximum occupancy
    int THREADS;
    int BLOCKS;
    hipOccupancyMaxPotentialBlockSize(&BLOCKS, &THREADS, bucket_module_sum_reduction_kernel_1, 0, 0);
    cout << "max threads is: " << THREADS << endl;
    cout << "max blocks is: " << BLOCKS << endl;

    // Read points
    context->pipp.read_jacobian_curve_points(j_points);
    context->pipp.read_scalars(d_scalars);

    unsigned bitsize = 254;
    unsigned c = 10;
    
    // Start timer
    using namespace std::chrono;
    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    context->pipp.initialize_buckets(d_scalars, j_points, bitsize, c, npoints);
    
    // End timer
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    std::cout << "It took me " << time_span.count() << " seconds." << endl;
}

/**
 * Perform naive MSM
 */ 
template <class A, class S, class J>
void msm_t<A, S, J>::pippenger_test(Context<bucket_t,point_t,scalar_t,affine_t> *context, size_t npoints, A *points) {
    point_t *j_points;
    S *d_scalars;
    J *final_result;
    point_t *result_jacobian;
    point_t *result_projective;

    // Allocate cuda memory 
    hipMallocManaged(&j_points, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&d_scalars, NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&result_jacobian, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&result_projective, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));
    hipMallocManaged(&final_result, 3 * NUM_POINTS * LIMBS * sizeof(uint64_t));

    // Read points and scalars
    context->pipp.read_jacobian_curve_points(j_points);
    context->pipp.read_scalars(d_scalars);

    test_double_add<<<1,4>>>(j_points, d_scalars, final_result);
    hipDeviceSynchronize();

    context->pipp.print_result(final_result);
}

}