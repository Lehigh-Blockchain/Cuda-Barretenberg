#include "kernel.cu"
#include <iostream>
#include <vector>

using namespace std;

namespace pippenger_common {
/***************************************** Function declerations for 'pippenger_t' class  *****************************************/

/**
 * Initialize cuda device and MSM parameters
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
pippenger_t<bucket_t, point_t, scalar_t, affine_t> 
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::initialize_msm(pippenger_t &config, size_t npoints) {
    CUDA_WRAPPER(hipSetDevice(config.device));

    hipDeviceProp_t prop;
    CUDA_WRAPPER(hipGetDeviceProperties(&prop, 0));

    // Set streaming multiprocessor count, where each SM contains N cuda cores
    sm_count = prop.multiProcessorCount;
    
    config.npoints = npoints;
    config.n = (npoints + WARP - 1) & ((size_t)0 - WARP);
    config.N = (sm_count * 256) / (NTHREADS * NWINS);

    return config;
}

/**
 * Calculate the amount of device storage required to store bases 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_bases(pippenger_t &config) {
    return config.n * sizeof(affine_t);
}

/**
 * Calculate the amount of device storage required to store scalars 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_scalars(pippenger_t &config) {
    return config.n * sizeof(scalar_t);
}

/**
 * Calculate the amount of device storage required to store buckets 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_buckets(pippenger_t &config) {    
    return config.N * sizeof(bucket_t) * NWINS * (1 << WBITS);
}

/**
 * Allocate device storage for bases
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_bases(pippenger_t &config) {
    return device_base_ptrs.allocate(get_size_bases(config));
}

/**
 * Allocate device storage for scalars
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_scalars(pippenger_t &config) {
    return device_scalar_ptrs.allocate(get_size_scalars(config));
}

/**
 * Allocate device storage for buckets
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_buckets(pippenger_t &config) {
    return device_bucket_ptrs.allocate(get_size_buckets(config));
}

/**
 * Return size of base pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_base_ptrs() {
    return device_base_ptrs.size();
}

/**
 * Return size of scalar pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_scalar_ptrs() {
    return device_scalar_ptrs.size();
}

/**
 * Return size of bucket pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_bucket_ptrs() {
    return device_bucket_ptrs.size();
}

/**
 * Transfer base points to GPU device
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::transfer_bases_to_device(
pippenger_t &config, size_t d_points_idx, const affine_t points[]) {    
    // Set cuda device and default stream
    CUDA_WRAPPER(hipSetDevice(config.device));

    hipStream_t stream = config.default_stream;

    // change to affine_t, along with device_base_ptrs
    affine_t *d_points = device_base_ptrs[d_points_idx];

    // hipMemcpyAsync() is non-blocking and asynchronous variant of hipMemcpy() that requires pinned memory.
    // Asynchronous transfers enable overalapping data transfers with kernel execution.
    CUDA_WRAPPER(hipMemcpyAsync(d_points, points, config.npoints * sizeof(*d_points), hipMemcpyHostToDevice, default_stream));
}

/**
 * Transfer scalars to GPU device
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::transfer_scalars_to_device(
pippenger_t &config, size_t d_scalars_idx, const scalar_t scalars[], hipStream_t aux_stream = nullptr) {
    // Set cuda device and auxilary stream
    hipSetDevice(config.device);
    hipStream_t stream = aux_stream;

    scalar_t *d_scalars = device_scalar_ptrs[d_scalars_idx];
    
    CUDA_WRAPPER(hipMemcpyAsync(d_scalars, scalars, config.npoints * sizeof(*d_scalars), hipMemcpyHostToDevice, stream));
}

/**
 * Result container
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::result_container_t 
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::result_container(pippenger_t &config) {
    result_container_t res(config.N);
    return res;
}

/**
 * Synchronize stream
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::synchronize_stream(pippenger_t &config) {
    CUDA_WRAPPER(hipSetDevice(config.device));
    CUDA_WRAPPER(hipStreamSynchronize(config.default_stream));
}

/**
 * Helper function  
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
template<typename... Types>
inline void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::launch_coop(
void(*f)(Types...), dim3 gridDim, dim3 blockDim, hipStream_t stream, Types... args) {
    void* va_args[sizeof...(args)] = { &args... };

    CUDA_WRAPPER(hipLaunchCooperativeKernel((const void*)f, gridDim, blockDim, va_args, 0, stream));
}

/**
 * Launch kernel
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::launch_kernel(
pippenger_t &config, size_t d_bases_idx, size_t d_scalar_idx, size_t d_buckets_idx) {
    // Set default stream
    hipStream_t stream = config.default_stream;

    // Pointers to malloced memory locations
    affine_t *d_points = device_base_ptrs[d_bases_idx];
    scalar_t *d_scalars = device_scalar_ptrs[d_scalar_idx];

    // Two-dimensional array of pointers to 'bucket_t' values with NWINS slices, each slice containing 1<<WBITS bucket_t pointers
    bucket_t (*d_buckets)[NWINS][1<<WBITS] = reinterpret_cast<decltype(d_buckets)>(device_bucket_ptrs[d_buckets_idx]);
    bucket_t (*d_none)[NWINS][NTHREADS][2] = nullptr;
    
    CUDA_WRAPPER(hipSetDevice(config.device));

    // Helper function that triggers the kernel launch
    launch_coop(
        pippenger, dim3(NWINS, config.N), NTHREADS, stream, (affine_t*)d_points, config.npoints, 
        (const scalar_t*)d_scalars, d_buckets, d_none
    );
}

/**
 * Read affine elliptic curve points from SRS
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
affine_t* pippenger_t<bucket_t, point_t, scalar_t, affine_t>::read_affine_curve_points() {
    auto reference_string = std::make_shared<gpu_waffle::FileReferenceString>(NUM_POINTS, "../srs_db");
    g1::affine_element* points = reference_string->get_monomials();

    return points;
}

/**
 * Read jacobian elliptic curve points from file
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
point_t* pippenger_t<bucket_t, point_t, scalar_t, affine_t>::read_jacobian_curve_points(point_t *points) {
    std::ifstream myfile ("../src/aztec/gpu/benchmark/tests/msm/points/curve_points.txt"); 

    if ( myfile.is_open() ) {   
        for (size_t i = 0; i < NUM_POINTS; i++) {
            for (size_t j = 0; j < 4; j++) {
                myfile >> points[i].x.data[j];
            }
            for (size_t y = 0; y < 4; y++) {
                myfile >> points[i].y.data[y];
            }
            for (size_t z = 0; z < 4; z++) {
                myfile >> points[i].z.data[z];
            }
        }   
    }
    myfile.close();
    
    return points;
} 

/**
 * Read scalars from scalar field
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
scalar_t* pippenger_t<bucket_t, point_t, scalar_t, affine_t>::read_scalars(scalar_t *scalars) {
    ifstream stream;
    stream.open("../src/aztec/gpu/msm/points/scalars.txt", ios::in);

    if ( stream.is_open() ) {   
        for (size_t i = 0; i < NUM_POINTS; i++) {
            for (size_t j = 0; j < 4; j++) {
                stream >> scalars[i].data[j];
            }
        }   
    }
    stream.close();
        
    return scalars;
}

/**
 * Print results
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::print_result(point_t *result) {
    for (int i = 0; i < LIMBS; i++) {
        printf("result is: %zu\n", result[0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result is: %zu\n", result[0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result is: %zu\n", result[0].z.data[i]);
    }
    printf("\n");
}

/**
 * Initialze buckets
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::initialize_buckets(
scalar_t *scalars, point_t *points, unsigned bitsize, unsigned c, size_t npoints) {
    // Calculate the number of windows 
    unsigned num_bucket_modules = bitsize / c; 
    if (bitsize / c) {  
        num_bucket_modules++;
    }

    point_t *buckets;
    size_t num_buckets = num_bucket_modules << c;
    // change memory size?
    hipMallocManaged(&buckets, num_buckets * 3 * 4 * sizeof(uint64_t));

    // Launch bucket initialization kernel
    unsigned NUM_THREADS = 1 << 10;
    unsigned NUM_BLOCKS = (num_buckets + NUM_THREADS - 1) / NUM_THREADS;

    // Need to see if initializing the buckets is even neccessary? Let's do it at the beggining anyways
    initialize_buckets_kernel<<<NUM_BLOCKS * 4, NUM_THREADS>>>(buckets); // 1024 * 26 operating on a single limb.
    hipDeviceSynchronize();

    cout << "b-bit scalar is: " << bitsize << endl;
    cout << "c-bit scalar is: " << c << endl;
    cout << "number of bucket modules is: " << num_bucket_modules << endl;
    cout << "number of bucket is: " << num_buckets << endl;
    cout << "number of blocks is: " << NUM_BLOCKS << endl;
    cout << "number of threads is: " << NUM_THREADS << endl;

    // cout << "bucket 0 is: " << buckets[26623].x.data[0] << endl;
    // cout << "bucket 0 is: " << buckets[26623].x.data[1] << endl;
    // cout << "bucket 0 is: " << buckets[26623].x.data[2] << endl;
    // cout << "bucket 0 is: " << buckets[26623].x.data[3] << endl;

    // cout << "bucket 0 is: " << buckets[26623].y.data[0] << endl;
    // cout << "bucket 0 is: " << buckets[26623].y.data[1] << endl;
    // cout << "bucket 0 is: " << buckets[26623].y.data[2] << endl;
    // cout << "bucket 0 is: " << buckets[26623].y.data[3] << endl;

    // cout << "bucket 0 is: " << buckets[26623].z.data[0] << endl;
    // cout << "bucket 0 is: " << buckets[26623].z.data[1] << endl;
    // cout << "bucket 0 is: " << buckets[26623].z.data[2] << endl;
    // cout << "bucket 0 is: " << buckets[26623].z.data[3] << endl;

    // Allocate memory for bucket and point indices
    unsigned *bucket_indices;
    unsigned *point_indices;
    hipMallocManaged(&bucket_indices, sizeof(unsigned) * npoints * (num_bucket_modules + 1));
    hipMallocManaged(&point_indices, sizeof(unsigned) * npoints * (num_bucket_modules + 1));

    // Split scalars into digits
    // NUM_THREADS * NUM_BLOCKS = NUM_BUCKETS --> each thread splits a single scalar into num_modules digits, each of size c. 
    unsigned NUM_BLOCKS_2 = ((npoints * (num_bucket_modules + 1)) + NUM_THREADS - 1) / NUM_THREADS;

    cout << "scalar 0 is: " << scalars[0].data[0] << endl;
    cout << "scalar 0 is: " << scalars[0].data[1] << endl;
    cout << "scalar 0 is: " << scalars[0].data[2] << endl;
    cout << "scalar 0 is: " << scalars[0].data[3] << endl;

    // Split sub-scalars into their own seperate buckets
    // not sure why we're passing pointers at an offset of npoints
    // Would be able to do this 4 coooperaitve groups if c = 16...look into it
    split_scalars_kernel<<<NUM_BLOCKS_2, NUM_THREADS>>>(bucket_indices + npoints, point_indices + npoints, scalars, npoints, num_bucket_modules, c);
    hipDeviceSynchronize();

    // cout << "UNSORTED: " << endl;
    // for (int i = 0; i < 30; i++) {
    //     cout << "bucket_indices: " << bucket_indices[1024 + i] << endl;
    // }

    // integrating CUB routines for things like offset calculations
    // sort indices from smallest to largest in order to group points that belong to same bucket
    unsigned *sort_indices_temp_storage{};
    size_t sort_indices_temp_storage_bytes; // need to figure out why this is 1, maybe a return value?

    // Determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + npoints, bucket_indices,
                                 point_indices + npoints, point_indices, npoints);

    cout << "sort_indices_temp_storage_bytes is: " << sort_indices_temp_storage_bytes << endl;
    
    // for (int i = 0; i < 30; i++) {
    //     cout << "point_indices: " << point_indices[1024 + i] << endl;
    // }

    hipMalloc(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);

    // perform the radix sort operation -- total number of sorts is num_bucket_modules. sorting arrays of bucket_indices and point_indices
    for (unsigned i = 0; i < num_bucket_modules; i++) {
        unsigned offset_out = i * npoints;
        unsigned offset_in = offset_out + npoints;
        // pffsets ensure each iteration operates on different set of points in input / output array
        // compressing inputs + offset --> offset in this method
        hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + offset_in,
                                    bucket_indices + offset_out, point_indices + offset_in, point_indices + offset_out, npoints);
    }
    hipFree(sort_indices_temp_storage);

    // for (int i = 0; i < 30; i++) {
    //     cout << "point_indices: " << point_indices[i] << endl;
    // }

    // cout << "SORTED: " << endl;
    // for (int i = 0; i < 30; i++) {
    //     cout << "bucket_indices: " << bucket_indices[1024 + i] << endl;
    // }

    // Next find the size of the buckets based on bucket_indices
    unsigned *single_bucket_indices;
    unsigned *bucket_sizes;
    unsigned *nof_buckets_to_compute;
    hipMalloc(&single_bucket_indices, sizeof(unsigned) * num_buckets);
    hipMalloc(&bucket_sizes, sizeof(unsigned) * num_buckets);
    hipMalloc(&nof_buckets_to_compute, sizeof(unsigned));
    unsigned *encode_temp_storage{};
    size_t encode_temp_storage_bytes = 0;

    // run length encoding computes a simple compressed representation of a sequence of input element
    hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                            nof_buckets_to_compute, num_bucket_modules * npoints);
    hipMalloc(&encode_temp_storage, encode_temp_storage_bytes);
    hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                            nof_buckets_to_compute, num_bucket_modules * npoints);
    hipFree(encode_temp_storage);

    //get offsets - where does each new bucket begin
    unsigned* bucket_offsets;
    hipMalloc(&bucket_offsets, sizeof(unsigned) * num_buckets);
    unsigned* offsets_temp_storage{};
    size_t offsets_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, num_buckets);
    hipMalloc(&offsets_temp_storage, offsets_temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, num_buckets);
    hipFree(offsets_temp_storage);

    // Launch accumulation kernel
    unsigned NUM_BLOCKS_3 = (num_buckets + NUM_THREADS - 1) / NUM_THREADS;
    accumulate_buckets_kernel<<<NUM_BLOCKS_3, NUM_THREADS * 4>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, points, num_buckets);
    hipDeviceSynchronize();

    // cout << "final_result is: " << buckets[0].x.data[0] << endl;
    // cout << "final_result is: " << buckets[0].x.data[1] << endl;
    // cout << "final_result is: " << buckets[0].x.data[2] << endl;
    // cout << "final_result is: " << buckets[0].x.data[3] << endl;

    // At this point we have n buckets and m bucket modules. Need to first sum up the n buckets per bucket module, and then
    // perform a final accumulation of the bucket modules. 

    point_t *final_result;
    hipMallocManaged(&final_result, num_bucket_modules * 3 * 4 * sizeof(uint64_t));
    // Need to optimize these kernel launch parameters?
    bucket_module_sum_reduction_kernel<<<1, num_bucket_modules * 4>>>(buckets, final_result, num_buckets, c);
    hipDeviceSynchronize();

    // Final accumulation kernel
    point_t *res;
    hipMallocManaged(&res, 3 * 4 * sizeof(uint64_t));
    final_accumulation_kernel<<<1,4>>>(final_result, res, num_bucket_modules, c);
    hipDeviceSynchronize();

    // free memory
}


/***************************************** Function declerations for 'device_ptr' class  *****************************************/

/**
 * Allocate memory using hipMalloc
 */
template <class T>
size_t device_ptr<T>::allocate(size_t bytes) {
    T* d_ptr;
    CUDA_WRAPPER(hipMalloc(&d_ptr, bytes));

    d_ptrs.push_back(d_ptr);
    return d_ptrs.size() - 1;
}

/**
 * Get size of d_ptrs vector
 */
template <class T>
size_t device_ptr<T>::size() {
    return d_ptrs.size();
}

/**
 * Operator overloading for device_ptr
 */
template <class T>
T* device_ptr<T>::operator[](size_t i) {
    if (i > d_ptrs.size() - 1) {
        cout << "Indexing error!" << endl;
        throw;
    }
    return d_ptrs[i];
}

}