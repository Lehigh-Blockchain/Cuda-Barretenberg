#include "kernel.cu"

using namespace std;

namespace pippenger_common {
/***************************************** Function declerations for 'pippenger_t' class  *****************************************/

/**
 * Initialize cuda device and MSM parameters
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
pippenger_t<bucket_t, point_t, scalar_t, affine_t> 
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::initialize_msm(pippenger_t &config, size_t npoints) {
    CUDA_WRAPPER(hipSetDevice(config.device));

    hipDeviceProp_t prop;
    CUDA_WRAPPER(hipGetDeviceProperties(&prop, 0));

    // Set streaming multiprocessor count, where each SM contains N cuda cores
    sm_count = prop.multiProcessorCount;
    
    config.npoints = npoints;
    config.n = (npoints + WARP - 1) & ((size_t)0 - WARP);
    config.N = (sm_count * 256) / (NTHREADS * NWINS);

    return config;
}

/**
 * Calculate the amount of device storage required to store bases 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_bases(pippenger_t &config) {
    return config.n * sizeof(point_t);
}

/**
 * Calculate the amount of device storage required to store scalars 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_scalars(pippenger_t &config) {
    return config.n * sizeof(scalar_t);
}

/**
 * Calculate the amount of device storage required to store buckets 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_buckets(pippenger_t &config) {    
    return config.N * sizeof(bucket_t) * NWINS * (1 << WBITS);
}

/**
 * Allocate device storage for bases
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_bases(pippenger_t &config) {
    return device_base_ptrs.allocate(get_size_bases(config));
}

/**
 * Allocate device storage for scalars
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_scalars(pippenger_t &config) {
    return device_scalar_ptrs.allocate(get_size_scalars(config));
}

/**
 * Allocate device storage for buckets
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_buckets(pippenger_t &config) {
    return device_bucket_ptrs.allocate(get_size_buckets(config));
}

/**
 * Return size of base pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_base_ptrs() {
    return device_base_ptrs.size();
}

/**
 * Return size of scalar pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_scalar_ptrs() {
    return device_scalar_ptrs.size();
}

/**
 * Return size of bucket pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_bucket_ptrs() {
    return device_bucket_ptrs.size();
}

/**
 * Transfer base points to GPU device
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::transfer_bases_to_device(
pippenger_t &config, size_t d_points_idx, const affine_t points[]) {    
    // Set cuda device and default stream
    CUDA_WRAPPER(hipSetDevice(config.device));

    hipStream_t stream = config.default_stream;

    // change to affine_t, along with device_base_ptrs
    point_t *d_points = device_base_ptrs[d_points_idx];

    // hipMemcpyAsync() is non-blocking and asynchronous variant of hipMemcpy() that requires pinned memory.
    // Asynchronous transfers enable overalapping data transfers with kernel execution.
    CUDA_WRAPPER(hipMemcpyAsync(d_points, points, config.npoints * sizeof(*d_points), hipMemcpyHostToDevice, default_stream));
}

/**
 * Transfer scalars to GPU device
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::transfer_scalars_to_device(
pippenger_t &config, size_t d_scalars_idx, const scalar_t scalars[], hipStream_t aux_stream = nullptr) {
    // Set cuda device and auxilary stream
    hipSetDevice(config.device);
    hipStream_t stream = aux_stream;

    scalar_t *d_scalars = device_scalar_ptrs[d_scalars_idx];
    
    CUDA_WRAPPER(hipMemcpyAsync(d_scalars, scalars, config.npoints * sizeof(*d_scalars), hipMemcpyHostToDevice, stream));
}

/**
 * Result container
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::result_container_t 
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::result_container(pippenger_t &config) {
    result_container_t res(config.N);
    return res;
}

/**
 * Synchronize stream
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::synchronize_stream(pippenger_t &config) {
    CUDA_WRAPPER(hipSetDevice(config.device));
    CUDA_WRAPPER(hipStreamSynchronize(config.default_stream));
}

/**
 * Helper function  
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
template<typename... Types>
inline void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::launch_coop(
void(*f)(Types...), dim3 gridDim, dim3 blockDim, hipStream_t stream, Types... args) {
    void* va_args[sizeof...(args)] = { &args... };

    CUDA_WRAPPER(hipLaunchCooperativeKernel((const void*)f, gridDim, blockDim, va_args, 0, stream));
}

template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::launch_kernel(
pippenger_t &config, size_t d_bases_idx, size_t d_scalar_idx, size_t d_buckets_idx) {
    // Set default stream
    hipStream_t stream = config.default_stream;

    // Pointers to malloced memory locations
    point_t *d_points = device_base_ptrs[d_bases_idx];
    scalar_t *d_scalars = device_scalar_ptrs[d_scalar_idx];

    // Two-dimensional array of pointers to 'bucket_t' values with NWINS slices, each slice containing 1<<WBITS bucket_t pointers
    bucket_t (*d_buckets)[NWINS][1<<WBITS] = reinterpret_cast<decltype(d_buckets)>(device_bucket_ptrs[d_buckets_idx]);
    bucket_t (*d_none)[NWINS][NTHREADS][2] = nullptr;
    
    CUDA_WRAPPER(hipSetDevice(config.device));

    // Helper function that triggers the kernel launch
    launch_coop(
        pippenger, dim3(NWINS, config.N), NTHREADS, stream, (const point_t*)d_points, config.npoints, 
        (const scalar_t*)d_scalars, d_buckets, d_none
    );
}

/***************************************** Function declerations for 'device_ptr' class  *****************************************/

/**
 * Allocate memory using hipMalloc
 */
template <class T>
size_t device_ptr<T>::allocate(size_t bytes) {
    T* d_ptr;
    CUDA_WRAPPER(hipMalloc(&d_ptr, bytes));

    d_ptrs.push_back(d_ptr);
    return d_ptrs.size() - 1;
}

/**
 * Get size of d_ptrs vector
 */
template <class T>
size_t device_ptr<T>::size() {
    return d_ptrs.size();
}

/**
 * Operator overloading for device_ptr
 */
template <class T>
T* device_ptr<T>::operator[](size_t i) {
    if (i > d_ptrs.size() - 1) {
        cout << "Indexing error!" << endl;
        throw;
    }
    return d_ptrs[i];
}

}