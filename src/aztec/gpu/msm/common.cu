#include "hip/hip_runtime.h"
#include "kernel.cu"
#include <iostream>
#include <vector>

namespace pippenger_common {
/***************************************** Function declerations for 'pippenger_t' class  *****************************************/

/**
 * Initialize cuda device and MSM parameters
 */
template <class point_t, class scalar_t>
pippenger_t<point_t, scalar_t> 
pippenger_t<point_t, scalar_t>::initialize_msm(pippenger_t &config, size_t npoints) {
    CUDA_WRAPPER(hipSetDevice(config.device));
    config.n = npoints;

    return config;
}

/**
 * Calculate the amount of device storage required to store bases 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_bases(pippenger_t &config) {
    return config.n * sizeof(point_t);
}

/**
 * Calculate the amount of device storage required to store scalars 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_scalars(pippenger_t &config) {
    // return config.n * sizeof(scalar_t);
    return config.n * LIMBS * sizeof(uint64_t);
}

/**
 * Calculate the amount of device storage required to store buckets 
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::get_size_buckets(pippenger_t &config) { 
    return (MODULES << C) * 3 * 4 * sizeof(uint64_t);
}

/**
 * Allocate device storage for bases
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::allocate_bases(pippenger_t &config) {
    return device_base_ptrs.allocate(get_size_bases(config));
}

/**
 * Allocate device storage for scalars
 */
template <class point_t, class scalar_t>
size_t pippenger_t<point_t, scalar_t>::allocate_scalars(pippenger_t &config) {
    return device_scalar_ptrs.allocate(get_size_scalars(config));
}

/**
 * Transfer base points to GPU device
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::transfer_bases_to_device(
pippenger_t &config, point_t *device_bases_ptrs, const point_t *points, hipStream_t aux_stream = nullptr) {    
    // Set cuda device and auxilary stream
    hipStream_t stream = (aux_stream == nullptr) ? default_stream : aux_stream;
    hipSetDevice(config.device);    

    // hipMemcpyAsync() is non-blocking and asynchronous variant of hipMemcpy() that requires pinned memory.
    hipMemcpyAsync(device_bases_ptrs, points, NUM_POINTS * LIMBS * sizeof(uint64_t), hipMemcpyHostToDevice, stream);
}

/**
 * Transfer scalars to GPU device
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::transfer_scalars_to_device(
pippenger_t &config, scalar_t *device_scalar_ptrs, fr *scalars, hipStream_t aux_stream = nullptr) {
    // Set cuda device and auxilary stream
    hipStream_t stream = (aux_stream == nullptr) ? default_stream : aux_stream;
    hipSetDevice(config.device);    

    hipMemcpyAsync(device_scalar_ptrs, scalars, NUM_POINTS * LIMBS * sizeof(uint64_t), hipMemcpyHostToDevice, stream);
}

/**
 * Synchronize stream
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::synchronize_stream(pippenger_t &config) {
    CUDA_WRAPPER(hipSetDevice(config.device));
    CUDA_WRAPPER(hipStreamSynchronize(config.default_stream));
}

/**
 * Launch kernel
 */
// template <class point_t, class scalar_t>
// void pippenger_t<point_t, scalar_t>::launch_kernel(
// pippenger_t &config, size_t d_bases_idx, size_t d_scalar_idx, size_t d_buckets_idx) {
//     // Set default stream
//     hipStream_t stream = config.default_stream;

//     // Pointers to malloced memory locations
//     point_t *d_points = device_base_ptrs[d_bases_idx];
//     scalar_t *d_scalars = device_scalar_ptrs[d_scalar_idx];
    
//     CUDA_WRAPPER(hipSetDevice(config.device));

//     g1_gpu::element *final_result;
//     hipMallocManaged(&final_result, NUM_POINTS * LIMBS * sizeof(uint64_t));    

//     hipDeviceSynchronize();

//     cout << "\nfinal_result is: " << final_result[0].x.data[0] << endl;
//     cout << "final_result is: " << final_result[0].x.data[1] << endl;
//     cout << "final_result is: " << final_result[0].x.data[2] << endl;
//     cout << "final_result is: " << final_result[0].x.data[3] << endl;
// }

/**
 * Print results
 */
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::print_result(g1_gpu::element *result_naive_msm, g1_gpu::element *result_bucket_method_msm) {
    for (int i = 0; i < 3; i++) {
        printf("result_naive_msm is: %zu\n", result_naive_msm[0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_naive_msm[0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_naive_msm is: %zu\n", result_naive_msm[0].z.data[i]);
    }
    printf("\n");
    for (int i = 0; i < 3; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_bucket_method_msm[0].x.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_bucket_method_msm[0].y.data[i]);
    }
    printf("\n");
    for (int i = 0; i < LIMBS; i++) {
        printf("result_bucket_method_msm is: %zu\n", result_bucket_method_msm[0].z.data[i]);
    }
}

/**
 * Verify double-and-add and pippenger's bucket method results
 * move to common.cuh file
 */ 
template <class point_t, class scalar_t>
void pippenger_t<point_t, scalar_t>::verify_result(point_t *result_1, point_t *result_2) {
    var *result;
    hipMallocManaged(&result, LIMBS * sizeof(uint64_t));
    
    comparator_kernel<<<1, 4>>>(result_1, result_2, result);
    hipDeviceSynchronize();

    assert (result[0] == 1);
    assert (result[1] == 1);
    assert (result[2] == 1);
    assert (result[3] == 1);

    cout << "MSM Result Verified!" << endl;
}

// /**
//  * Execute bucket method 
//  */
template <class point_t, class scalar_t>
point_t* pippenger_t<point_t, scalar_t>::execute_bucket_method(
pippenger_t &config, scalar_t *scalars, point_t *points, unsigned bitsize, unsigned c, size_t npoints) {
    unsigned num_bucket_modules = bitsize / c; 
    if (bitsize % c) {  
        num_bucket_modules++;
    }
    size_t num_buckets = num_bucket_modules << c; // 1024 * 26 or 65536 * 16 = 1M buckets


    point_t *buckets;
    // change memory size?
    hipMallocManaged(&buckets, num_buckets * 3 * 4 * sizeof(uint64_t));

    // Launch bucket initialization kernel
    unsigned NUM_THREADS = 1 << 10; // max number of threads
    unsigned NUM_BLOCKS = (num_buckets + NUM_THREADS - 1) / NUM_THREADS; // max number of blocks

    // where blocks * threads = total numbrt of buckets

    // Need to see if initializing the buckets is even neccessary? Let's do it at the beggining anyways for now...
    // and later incorperate it into the kernel with "pipelining loads and compute" as depaul put it.
    // Each thread operates on a single scalar of size num_bucket_modules windows,
    // But there's a problem here. There's a mismatch between the number of threads being launched to represent the 
    // number of buckets, and the total number of scalars that need to be partitioned into these buckets. I think the
    // buckets array needs to be a 2D array to account for this? Will change after...
    // It will need to support # scalars * 16, which may exceed the bucket count. Therefore the total number of buckets will
    // stay the same, but the indexing will change...2D instead.
    initialize_buckets_kernel<<<NUM_BLOCKS * 4, NUM_THREADS>>>(buckets); 
    // hipDeviceSynchronize();

    // hipMallocManaged(&buckets, num_buckets * 3 * 4 * sizeof(uint64_t));
    // // convert affine to jacobian coordinates
    // af<<<NUM_BLOCKS * 4, NUM_THREADS>>>(points, points); 

    cout << "b-bit scalar is: " << bitsize << endl;
    cout << "c-bit scalar is: " << c << endl;
    cout << "number of bucket modules is: " << num_bucket_modules << endl;
    cout << "number of bucket is: " << num_buckets << endl;
    cout << "number of blocks is: " << NUM_BLOCKS << endl;
    cout << "number of threads is: " << NUM_THREADS << endl;
    
    // cout << "bucket 0 is: " << buckets[1048575].y.data[0] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].x.data[1] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].x.data[2] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].x.data[3] << endl;

    // cout << "bucket 0 is: " << buckets[1048575].y.data[0] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].y.data[1] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].y.data[2] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].y.data[3] << endl;

    // cout << "bucket 0 is: " << buckets[1048575].z.data[0] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].z.data[1] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].z.data[2] << endl;
    // cout << "bucket 0 is: " << buckets[1048575].z.data[3] << endl;

    // After here, we have 2^16 windows, with each scalar represented by 16 bits. Each window will have 2^16 buckets,
    // and we'll have 2^16 * 16 total buckets. 

    // Allocate memory for bucket and point indices
    // need to understand these launch parameters
    unsigned *bucket_indices;
    unsigned *point_indices;
    hipMallocManaged(&bucket_indices, sizeof(unsigned) * npoints * (num_bucket_modules + 1));
    hipMallocManaged(&point_indices, sizeof(unsigned) * npoints * (num_bucket_modules + 1));

    // Split scalars into digits
    // NUM_THREADS * NUM_BLOCKS = NUM_BUCKETS --> each thread splits a single scalar into num_modules digits, each of size c. 
    // unsigned NUM_BLOCKS_2 = ((npoints * (num_bucket_modules + 1)) + NUM_THREADS - 1) / NUM_THREADS;
    // unsigned NUM_BLOCKS_2 = ((num_buckets + NUM_THREADS - 1) / NUM_THREADS);
    unsigned NUM_BLOCKS_2 = NUM_POINTS / NUM_THREADS;
    

    cout << "npoints is: " << npoints << endl;
    cout << "NUM_THREADS is: " << NUM_THREADS << endl;
    cout << "NUM_BLOCKS_2 is: " << NUM_BLOCKS_2 << endl;

    // cout << "scalar 0 is: " << scalars[0].data[0] << endl;
    // cout << "scalar 0 is: " << scalars[0].data[1] << endl;
    // cout << "scalar 0 is: " << scalars[0].data[2] << endl;
    // cout << "scalar 0 is: " << scalars[0].data[3] << endl;
    // cout << "points is: " << points[0].x.data[0] << endl;
    // cout << "points is: " << points[0].x.data[1] << endl;
    // cout << "points is: " << points[0].x.data[2] << endl;
    // cout << "points is: " << points[0].x.data[3] << endl;

    // Split sub-scalars into their own seperate buckets
    // not sure why we're passing pointers at an offset of npoints
    // Would be able to do this 4 coooperaitve groups if c = 16...look into it
    // Why do we have 27 blocks here?? 27 * 1024 = 27648 -- maybe because of the offset?
    // i think I can reduce this to 26 blocks

    // 1024 * 17 = 17408 threads to split scalars, but have more than 17408 scalars?
    // What if you have more scalars than buckets, which is the case here. Previously, there were less
    // scalars (1024) than the total number of buckets (27648).
    
    // Each thread will handle splitting it's own scalar into sub-scalars, and placing them into buckets. 
    // split_scalars_kernel<<<NUM_BLOCKS_2, NUM_THREADS>>>(bucket_indices + npoints, point_indices + npoints, scalars, npoints, num_bucket_modules, c);
    // ********* This scalar value for launch parameters will also need to be changed instead of being hardcoded! ******************
    split_scalars_kernel<<<NUM_BLOCKS_2, NUM_THREADS>>>(bucket_indices + npoints, point_indices + npoints, scalars, npoints, num_bucket_modules, c);
    hipDeviceSynchronize();

    // integrating CUB routines for things like offset calculations
    // sort indices from smallest to largest in order to group points that belong to same bucket
    unsigned *sort_indices_temp_storage{};
    size_t sort_indices_temp_storage_bytes; // need to figure out why this is 1, maybe a return value?

    // Determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + npoints, bucket_indices,
                                 point_indices + npoints, point_indices, npoints);

    // cout << "sort_indices_temp_storage_bytes is: " << sort_indices_temp_storage_bytes << endl;

    hipMallocManaged(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);

    // perform the radix sort operation -- total number of sorts is num_bucket_modules. sorting arrays of bucket_indices and point_indices
    for (unsigned i = 0; i < num_bucket_modules; i++) {
        unsigned offset_out = i * npoints;
        unsigned offset_in = offset_out + npoints;
        // pffsets ensure each iteration operates on different set of points in input / output array
        // compressing inputs + offset --> offset in this method
        hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + offset_in,
                                    bucket_indices + offset_out, point_indices + offset_in, point_indices + offset_out, npoints);
    }
    // hipFree(sort_indices_temp_storage);

    // we've eliminated the offset here
    // cout << "SORTED: " << endl;
    // for (int i = 0; i < 1024; i++) {
    //     cout << "index is: " << i << endl;
    //     cout << "bucket_indices: " << bucket_indices[i] << endl;
    // }
    // exit(0);

    // Next find the size of the buckets based on bucket_indices
    unsigned *single_bucket_indices;
    unsigned *bucket_sizes;
    unsigned *nof_buckets_to_compute;
    // change this from unifiedc memory to hipMalloc
    hipMallocManaged(&single_bucket_indices, sizeof(unsigned) * num_buckets);
    hipMallocManaged(&bucket_sizes, sizeof(unsigned) * num_buckets);
    hipMallocManaged(&nof_buckets_to_compute, sizeof(unsigned));
   
    void *encode_temp_storage = NULL;
    size_t encode_temp_storage_bytes = 0;

    // run length encoding computes a simple compressed representation of a sequence of input element
    // this returns the unique bucket #, number of buckets in each, and the total number of unique buckets
    hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                            nof_buckets_to_compute, num_bucket_modules * npoints);
    hipMallocManaged(&encode_temp_storage, encode_temp_storage_bytes);
    hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                            nof_buckets_to_compute, num_bucket_modules * npoints);
    hipFree(encode_temp_storage);


    cout << "nof_buckets_to_compute is: " << nof_buckets_to_compute[0] << endl;

    // int sum = 0;
    // for (int i = 0; i < nof_buckets_to_compute[0]; i++) {
    //     sum = sum + bucket_sizes[i];
    // }

    // cout << "sum is: " << sum << endl;

    // int temp = 0;
    // for (int i = 0; i < nof_buckets_to_compute[0]; i++) {
    //     cout << "count is: " << temp << endl;
    //     cout << "single_bucket_indices: " << single_bucket_indices[i] << endl;
    //     temp++;
    // }

    // int temp = 0;
    // for (int i = 0; i < num_buckets; i++) {
    //     if (single_bucket_indices[i] != 0) {
    //         temp++;
    //     }
    // }
    // cout << "temp is: " << temp << endl;

    // for (int i = 0; i < num_buckets; i++) {
    //     if (bucket_sizes[i] == 2) {
    //         cout << "!!!!!!!!!\n" << endl;
    //     } 
    // }

    //get offsets - where does each new bucket begin
    unsigned* bucket_offsets;
    hipMallocManaged(&bucket_offsets, sizeof(unsigned) * num_buckets);
    // unsigned* offsets_temp_storage{};
    void *offsets_temp_storage = NULL;
    size_t offsets_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, num_buckets);
    hipMallocManaged(&offsets_temp_storage, offsets_temp_storage_bytes);
    hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, num_buckets);
    hipFree(offsets_temp_storage);

    // cout << "num_buckets is: " << num_buckets << endl;

    // cout << "bucekt size is: " << bucket_sizes[5] << endl;

    // This is correct
    // for (int i = 0; i < num_buckets; i++) {
    //     if (bucket_sizes[i] != 0) {
    //         cout << "index is: " << i << endl;
    //         cout << "bucket size is: " << bucket_sizes[i] << endl;
    //     }
    // }

    // exit(0);

    // Launch accumulation kernel
    // 512 and 208 respectively
    //Need to adjusge kernel para,meters to reduce overhead
    unsigned NUM_THREADS_3 = 1 << 7;
    unsigned NUM_BLOCKS_3 = ((num_buckets + NUM_THREADS_3 - 1) / NUM_THREADS_3) * 4;

    cout << "NUM_THREADS_3 is: " << NUM_THREADS_3 << endl;
    cout << "NUM_BLOCKS_3 is: " << NUM_BLOCKS_3 << endl;
    
    //  // CUDA Event API
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // hipEventRecord(start);

    // Calculate maximum occupancy
    // int THREADS;
    // int BLOCKS;
    // hipOccupancyMaxPotentialBlockSize(&BLOCKS, &THREADS, accumulate_buckets_kernel, 0, 0);
    // cout << "NUM_POINTS is: " << NUM_POINTS << endl;
    // cout << "max threads is: " << THREADS << endl;
    // cout << "max blocks is: " << BLOCKS << endl;

    // The kernel launch parameters need to be changed!
    // What’s the primary reason why the max occupancy (threads + blocks) of a kernel is much lower than the 
    // theoretical occupancy (# SMs * max threads / SM)? I have a complex kernel function that exceeds all the 
    // registers past 512 threads / block. I have to lower the thread count per block and launch more blocks as a 
    // result. But i’m hitting an upper-block limit for this kernel. Is the only solution here splitting up the problem
    // between multiple seperate kernel launches? And if so, what’s the best approach? The naive solution is moving from
    // single P100 to A10. Update: the issue was an if statement in the kernel, which solved the problem. 
    // accumulate_buckets_kernel<<<NUM_BLOCKS_3, NUM_THREADS_3>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, points, num_buckets);
    accumulate_buckets_kernel<<<NUM_BLOCKS_3, NUM_THREADS_3>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, points, num_buckets);
    hipDeviceSynchronize();
    
    // hipEventRecord(stop);

    // // Synchronization barrier that blocks CPU execution until the specified event is recorded
    // hipEventSynchronize(stop);

    // // Calculate duraion of execution time 
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // cout << "Time taken by function: " << milliseconds << " milliseconds\n" << endl; 

    // exit(0);

    // hipDeviceSynchronize();

    // // End timer
    // high_resolution_clock::time_point t2 = high_resolution_clock::now();
    // duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    // std::cout << "It took me " << time_span.count() << " seconds." << endl;
    // exit(0);



    // Need to reconcile the lauch paramerters here............ 

    unsigned *bucket_index;
    hipMallocManaged(&bucket_index, num_bucket_modules * sizeof(unsigned));
    // int count = 0;
    // for (int i = 0; i < num_buckets; i++) {
    //     if (buckets[i].x.data[0] != 0) {
    //         cout << "bucket index is: " << i << endl;
    //         bucket_index[count] = i;
    //         cout << "bucket_index[count] is; " << bucket_index[count] << endl;
    //         cout << "x: " << "{ " << buckets[i].x.data[0] << "," << buckets[i].x.data[1] << "," << buckets[i].x.data[2] << "," << buckets[i].x.data[3] << " }; "  
    //         << " y: " << "{ " << buckets[i].y.data[0] << "," << buckets[i].y.data[1] << "," << buckets[i].y.data[2] << "," << buckets[i].y.data[3] << " }; "  
    //         << " z: " << "{ " << buckets[i].z.data[0] << "," << buckets[i].z.data[1] << "," << buckets[i].z.data[2] << "," << buckets[i].z.data[3] << " }; " << endl;
    //         count++;
    //     }
    // }
    // cout << "count is: " << count << endl;
    // cout << "??????????????????????????????????????????????\n" << endl;

    // for (int i = 0; i < num_bucket_modules; i++) {
    //     cout << "bucket_index is: " << bucket_index[i] << endl;
    // }
    hipDeviceSynchronize();


    // At this point we have n buckets and m bucket modules. Need to first sum up the n buckets per bucket module, and then
    // perform a final accumulation of the bucket modules. Launch 4 threads per bucket module.
    
    // Define parameters 
    // unsigned M = 1;
    // unsigned U = (1 << c) / M; // Need to add some offset here and in for loop

    // cout << "U is: " << U << endl;

    // point_t *S;
    // point_t *G;
    // point_t *result;
    point_t *final_sum;
    // hipMallocManaged(&S, num_bucket_modules * M * 3 * 4 * sizeof(uint64_t));
    // hipMallocManaged(&G, num_bucket_modules * M * 3 * 4 * sizeof(uint64_t));
    // hipMallocManaged(&result, num_bucket_modules * 3 * 4 * sizeof(uint64_t));
    hipMallocManaged(&final_sum, num_bucket_modules * 3 * 4 * sizeof(uint64_t));
    // or change kernel parameters to num_bucket_modules, 4
    // need to look into replacing this with known sum reduction techniques, since it dominates 90% of the runtime
    // Here we're launching 256 with 8 blocks, where each group of 4 threads handles adding 2 points.
    // so 256 / 4 = 64 x 2 x 8 = 1024 points 
    // kernels in default stream are invoked sequentially
    // These parameters will need to be more generalized below
    bucket_module_sum_reduction_lernel_0<<<26, 4>>>(buckets, final_sum, c);
    hipDeviceSynchronize();

    // cout << "PRINTING bucket_module_sum_reduction_lernel_0: " << endl;
    // for (int i = 0; i < 26; i++) {
    //     for (int j = 0; j < LIMBS; j++) {
    //         printf("result is: %zu\n", final_sum[i].x.data[j]);
    //     }
    //     printf("\n");
    //     for (int y = 0; y < LIMBS; y++) {
    //         printf("result is: %zu\n", final_sum[i].y.data[y]);
    //     }
    //     printf("\n");
    //     for (int z = 0; z < LIMBS; z++) {
    //         printf("result is: %zu\n", final_sum[i].z.data[z]);
    //     }
    //     printf("!!!!!!!!!\n");
    // }
    // printf("\n");

    // hipSetDevice(0);
    // size_t free_device_mem = 0;
    // size_t total_device_mem = 0;
    // hipMemGetInfo(&free_device_mem, &total_device_mem);
    // printf("Currently available amount of device memory: %zu bytes\n", free_device_mem);
    // printf("Total amount of device memory: %zu bytes\n", total_device_mem);
    
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );
    
    // bucket_module_sum_reduction_kernel_1<<<26, 4>>>(buckets, S, G, M, U);
    // hipDeviceSynchronize();

    // cout << "<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>" << endl;
    // cout << "PRINTING bucket_module_sum_reduction_kernel_1: " << endl;
    // for (int i = 0; i < 26; i++) {
    //     for (int j = 0; j < LIMBS; j++) {
    //         printf("result is: %zu\n", G[i].x.data[j]);
    //     }
    //     printf("\n");
    //     for (int y = 0; y < LIMBS; y++) {
    //         printf("result is: %zu\n", G[i].y.data[y]);
    //     }
    //     printf("\n");
    //     for (int z = 0; z < LIMBS; z++) {
    //         printf("result is: %zu\n", G[i].z.data[z]);
    //     }
    //     printf("!!!!!!!!\n");
    // }
    // printf("\n");

    // launch where each group of 4 threads will handle a seperate window k and all M segments within a window K -- problem is small
    // bucket_module_sum_reduction_kernel_2<<<26, 4>>>(result, S, G, M, U);
    // hipDeviceSynchronize();

    // for (int i = 0; i < 1; i++) {
    //     for (int j = 0; j < LIMBS; j++) {
    //         printf("result is: %zu\n", result[i].x.data[j]);
    //     }
    //     printf("\n");
    //     for (int j = 0; j < LIMBS; j++) {
    //         printf("result is: %zu\n", result[i].y.data[j]);
    //     }
    //     printf("\n");
    //     for (int j = 0; j < LIMBS; j++) {
    //         printf("result is: %zu\n", result[i].z.data[j]);
    //     }
    //     printf("!!!!!!!!\n");
    // }
    // printf("\n");
    // printf(">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>\n");

    // exit(0);

    printf("FINAL ACCUMULATION TEST\n");

    // This is still unified memory, which might impact performance when paired with hipDeviceSynchronize?
    point_t *res;
    hipMallocManaged(&res, 3 * 4 * sizeof(uint64_t));
    // change points to partial sums
    final_accumulation_kernel<<<1,4>>>(final_sum, res, num_bucket_modules, c);
    // final_accumulation_kernel_test<<<1,4>>>(points, res, num_bucket_modules, c);
    hipDeviceSynchronize();

    cout << "final_accumulation_kernel_test is: " << res[0].x.data[0] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].x.data[1] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].x.data[2] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].x.data[3] << endl;
    cout << endl;

    cout << "final_accumulation_kernel_test is: " << res[0].y.data[0] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].y.data[1] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].y.data[2] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].y.data[3] << endl;
    cout << endl;

    cout << "final_accumulation_kernel_test is: " << res[0].z.data[0] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].z.data[1] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].z.data[2] << endl;
    cout << "final_accumulation_kernel_test is: " << res[0].z.data[3] << endl;
    cout << endl;

    return res;

    // free memory
}

/***************************************** Function declerations for 'device_ptr' class  *****************************************/

/**
 * Allocate memory using hipHostMalloc
 */
template <class T>
size_t device_ptr<T>::allocate(size_t bytes) {
    T* d_ptr;
    CUDA_WRAPPER(hipHostMalloc(&d_ptr, bytes));
    // CUDA_WRAPPER(hipHostMalloc(&context->h_scalars, context->pipp.get_size_scalars(context->pipp)));

    d_ptrs.push_back(d_ptr);
    return d_ptrs.size() - 1;
}

/**
 * Get size of d_ptrs vector
 */
template <class T>
size_t device_ptr<T>::size() {
    return d_ptrs.size();
}

/**
 * Operator overloading for device_ptr
 */
template <class T>
T* device_ptr<T>::operator[](size_t i) {
    if (i > d_ptrs.size() - 1) {
        cout << "Indexing error!" << endl;
        throw;
    }
    return d_ptrs[i];
}

}