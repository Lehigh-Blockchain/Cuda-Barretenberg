#include "./common.cuh"

using namespace std;

namespace pippenger_common {
/***************************************** Function declerations for 'pippenger_t' class  *****************************************/

/**
 * Initialize cuda device and MSM parameters
 */
template <>
pipp_t pipp_t::initialize_msm(size_t npoints) {
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    sm_count = prop.multiProcessorCount;

    pipp_t config;
    config.npoints = npoints;
    config.n = (npoints + WARP - 1) & ((size_t)0 - WARP);
    config.N = (sm_count * 256) / (NTHREADS * NWINS);

    return config;
}

/**
 * Calculate the amount of storage neccessary to store bases 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_bases(pippenger_t &config) {
    return config.n * sizeof(point_t);
}

/**
 * Calculate the amount of storage neccessary to store scalars 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_scalars(pippenger_t &config) {
    return config.n * sizeof(scalar_t);
}

/**
 * Calculate the amount of storage neccessary to store buckets 
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::get_size_buckets(pippenger_t &config) {
    return config.N * sizeof(bucket_t) * NWINS * (1 << WBITS);
}

/**
 * Allocate device storage for bases
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_bases(pippenger_t &config) {
    return device_base_ptrs.allocate(get_size_bases(config));
}

/**
 * Allocate device storage for scalars
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_scalars(pippenger_t &config) {
    return device_scalar_ptrs.allocate(get_size_scalars(config));
}

/**
 * Allocate device storage for buckets
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::allocate_buckets(pippenger_t &config) {
    return device_bucket_ptrs.allocate(get_size_buckets(config));
}

/**
 * Return size of base pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_base_ptrs() {
    return device_base_ptrs.size();
}

/**
 * Return size of scalar pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_scalar_ptrs() {
    return device_scalar_ptrs.size();
}

/**
 * Return size of bucket pointers
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
size_t pippenger_t<bucket_t, point_t, scalar_t, affine_t>::num_bucket_ptrs() {
    return device_bucket_ptrs.size();
}

/**
 * Transfer base points to GPU device
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::transfer_bases_to_device(
pippenger_t &config, size_t d_points_idx, const affine_t points[], size_t ffi_affine_sz) {
    hipSetDevice(0);
    hipStream_t stream = 0; // default stream
    affine_t *d_points = device_base_ptrs[d_points_idx];

    if (ffi_affine_sz != sizeof(*d_points)) {
        hipError_t status = hipMemcpy2DAsync(
            d_points, sizeof(*d_points), points, ffi_affine_sz,ffi_affine_sz, config.npoints,hipMemcpyHostToDevice, stream);
        if (status != hipSuccess) {
            printf("Error copying bases to device\n");
        }
    }
    else {
        hipError_t status = hipMemcpyAsync(d_points, points, config.npoints * sizeof(*d_points), hipMemcpyHostToDevice, stream);
        if (status != hipSuccess) {
            printf("Error copying bases to device\n");
        } 
    }
}

/**
 * Transfer scalars to GPU device
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
void pippenger_t<bucket_t, point_t, scalar_t, affine_t>::transfer_scalars_to_device(
pippenger_t &config, size_t d_scalars_idx, const scalar_t scalars[], hipStream_t s) {
    hipSetDevice(0);
    hipStream_t stream = 0; // default stream
    scalar_t *d_scalars = device_scalar_ptrs[d_scalars_idx];

    hipError_t status = hipMemcpy2DAsync(d_scalars, scalars, config.npoints*sizeof(*d_scalars), hipMemcpyHostToDevice, stream);
    if (status != hipSuccess) {
        printf("Error copying bases to device\n");
    }
}

/**
 * Result container
 */
template <class bucket_t, class point_t, class scalar_t, class affine_t>
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::result_container_t 
pippenger_t<bucket_t, point_t, scalar_t, affine_t>::result_container(pippenger_t &config) {
    result_container_t res(config.N);
    return res;
}

/***************************************** Function declerations for 'device_ptr' class  *****************************************/

/**
 * Allocate memory using hipMalloc
 */
template <class T>
size_t device_ptr<T>::allocate(size_t bytes) {
    T* d_ptr;
    hipMalloc(&d_ptr, bytes);
    d_ptrs.push_back(d_ptr);
    return d_ptrs.size() - 1;
}

/**
 * Get size of d_ptrs vector
 */
template <class T>
size_t device_ptr<T>::size() {
    return d_ptrs.size();
}

/**
 * Operator overloading for device_ptr
 */
template <class T>
T* device_ptr<T>::operator[](size_t i) {
    if (i > d_ptrs.size() - 1) {
        cout << hipErrorInvalidDevicePointer << endl;
    }
    return d_ptrs[i];
}

}
