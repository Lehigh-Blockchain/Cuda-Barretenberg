#include "hip/hip_runtime.h"
#include "group.cu"

#include <assert.h> 
#include <iostream>
#include <algorithm>
#include <memory>
#include <fstream>
#include <string>

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg;

// Kernel launch parameters
static constexpr size_t BLOCKS = 256;
static constexpr size_t THREADS = 256;
static constexpr size_t POINTS = 1 << 16;

/* -------------------------- Kernel Functions For Finite Field Tests ---------------------------------------------- */

// Sum reduction with warp divergence
__global__ void sum_reduction(int *v, int *v_r) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction in shared memory
    __shared__ int partial_sum[256];

    // Partial_sum array is being used to accumulate partial sums
    partial_sum[threadIdx.x] = v[tid];

    // Sychronization barrier
    __syncthreads();
    
    // Warp divergence to determine active threads
    for (int s = 1; s < blockDim.x; s *= 2) {
        if (threadIdx.x % (2 * s) == 0) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Accumulate result into current block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

/* -------------------------- Helper Functions ---------------------------------------------- */

/**
 * Print finite field tests
 */
void print_field_tests(var *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print results for each limb
    printf("result[0] is: %zu\n", result[0]);
    printf("result[1] is: %zu\n", result[1]);
    printf("result[2] is: %zu\n", result[2]);
    printf("result[3] is: %zu\n\n", result[3]);
}

/* -------------------------- Executing Initialization and Workload Kernels ---------------------------------------------- */

// Execute kernel with vector of finite field elements
void execute_kernels(var *a, var *b, var *c, var *d, var *result, var *res_x, var *res_y, var *res_z) {    
    size_t bytes = POINTS * sizeof(int);

    // Allocate dynamic memory
    int *h_v, *h_v_r, *d_v, *d_v_r;
    h_v = (int *)malloc(POINTS * sizeof(int));
    h_v_r = (int *)malloc(POINTS * sizeof(int));
    hipMalloc(&d_v, POINTS * sizeof(int));
    hipMalloc(&d_v_r, POINTS * sizeof(int));

    // Populate array
    for (int i = 0; i < POINTS; i++) {
        h_v[i] = 1;
    }

    // Copy array to device
    hipMemcpy(d_v, h_v, POINTS * sizeof(int), hipMemcpyHostToDevice);

    // Grid size
    int GRID_SIZE = (int)ceil(POINTS / 256);

    // Call kernel
    sum_reduction<<<GRID_SIZE, 256>>>(d_v, d_v_r);

    // Final accumulation
    sum_reduction<<<1, GRID_SIZE>>>(d_v_r, d_v_r);

    // Copy results to host
    hipMemcpy(h_v_r, d_v_r, POINTS * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Accumulated result is: " << h_v_r[0] << endl;
}

/* -------------------------- Main Entry Function ---------------------------------------------- */

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define pointers to uint64_t type
    var *a, *b, *c, *d, *result, *res_x, *res_y, *res_z;

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS * sizeof(var));
    hipMallocManaged(&b, LIMBS * sizeof(var));
    hipMallocManaged(&c, LIMBS * sizeof(var));
    hipMallocManaged(&d, LIMBS * sizeof(var));
    hipMallocManaged(&result, LIMBS * sizeof(var));
    hipMallocManaged(&res_x, LIMBS * sizeof(var));
    hipMallocManaged(&res_y, LIMBS * sizeof(var));
    hipMallocManaged(&res_z, LIMBS * sizeof(var));

    // Execute kernel functions
    execute_kernels(a, b, c, d, result, res_x, res_y, res_z);

    // Successfull execution of unit tests
    cout << "******* All 'MSM' unit tests passed! **********" << endl;

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(result);

    cout << "Completed sucessfully!" << endl;

    return 0;
}