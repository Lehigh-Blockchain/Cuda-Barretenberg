#include "hip/hip_runtime.h"
#include "group.cu"

#include <assert.h> 
#include <iostream>
#include <algorithm>
#include <memory>
#include <fstream>
#include <string>
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace std::chrono;
using namespace gpu_barretenberg;
using namespace cooperative_groups;

// Kernel launch parameters
static constexpr size_t BLOCKS = 256;
static constexpr size_t THREADS = 256;
static constexpr size_t POINTS = 1 << 16;

/* -------------------------- Kernel Functions ---------------------------------------------- */

/* -------------------------- Kernel 1 ---------------------------------------------- */

// Sum reduction with warp divergence
__global__ void sum_reduction_1(int *v, int *v_r) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction in shared memory
    __shared__ int partial_sum[256];

    // Partial_sum array is being used to accumulate partial sums
    partial_sum[threadIdx.x] = v[tid];

    // Sychronization barrier
    __syncthreads();
    
    // Warp divergence to determine active threads based on stride
    for (int s = 1; s < blockDim.x; s *= 2) {
        if (threadIdx.x % (2 * s) == 0) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Accumulate result into current block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

/* -------------------------- Kernel 2 ---------------------------------------------- */

// Sum reduction using sequential threads (eliminating warp divergence and modulo operation). 
// This performs ~2x compared to sum_reduction_1. 
__global__ void sum_reduction_2(int *v, int *v_r) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction in shared memory
    __shared__ int partial_sum[256];

    // Partial_sum array is being used to accumulate partial sums
    partial_sum[threadIdx.x] = v[tid];

    // Sychronization barrier
    __syncthreads();
    
    for (int s = 1; s < blockDim.x; s *= 2) {
        // Change the indexing to be sequential threads (i.e. divide threads into groups)
        int index = 2 * s * threadIdx.x;

        // Each thread does work unless the idex goes off the block
        if (index < blockDim.x) {
            partial_sum[index] += partial_sum[index + s];
        }
        __syncthreads();
    }

    // Accumulate result into current block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

/* -------------------------- Kernel 3 ---------------------------------------------- */

// Contiguous memory access instead of strided access, avoiding shared memory bank conflicts.
// Bank conflicts arise because of some specific access pattern of data in shared memory.
__global__ void sum_reduction_3(int *v, int *v_r) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction in shared memory
    __shared__ int partial_sum[256];

    // Partial_sum array is being used to accumulate partial sums
    partial_sum[threadIdx.x] = v[tid];

    // Sychronization barrier
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        // Each thread does work unless the index goes off the block
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Accumulate result into current block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

/* -------------------------- Kernel 4 ---------------------------------------------- */

// Half the threads are idle after the first iteration. Instead,
// launch half the number of threads and pack more work. 
__global__ void sum_reduction_4(int *v, int *v_r) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction in shared memory
    __shared__ int partial_sum[256];

    // Load elements and perform first pass of the reduction,
    // Scale i since vector is 2x as long as the number of threads
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Partial_sum array is being used to accumulate partial sums
    partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];

    // Sychronization barrier
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        // Each thread does work unless the idex goes off the block
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Accumulate result into current block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

/* -------------------------- Kernel 4 ---------------------------------------------- */

// Used for last iteration to save useless work
// 'volatile' to prevent caching in registers (compiler optimization)
__device__ void warpReduce(volatile int *shared_mem_ptr, int t) {
    shared_mem_ptr[t] += shared_mem_ptr[t + 32];
    shared_mem_ptr[t] += shared_mem_ptr[t + 16];
    shared_mem_ptr[t] += shared_mem_ptr[t + 8];
    shared_mem_ptr[t] += shared_mem_ptr[t + 4];
    shared_mem_ptr[t] += shared_mem_ptr[t + 2];
    shared_mem_ptr[t] += shared_mem_ptr[t + 1];
}

// Warp reduce with loop unrolling
__global__ void sum_reduction_5(int *v, int *v_r) { 
    // Global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction in shared memory
    __shared__ int partial_sum[256];

    // Load elements and perform first pass of the reduction,
    // Scale i since vector is 2x as long as the number of threads
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Partial_sum array is being used to accumulate partial sums
    partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];

    // Sychronization barrier
    __syncthreads();
    
    // Do all iterations until reaching the last warp, otherwise
    // results in a lot of useless checks
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        // Each thread does work unless it is further than the stride
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        warpReduce(partial_sum, threadIdx.x);
    }

    // Accumulate result into current block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

/* -------------------------- Helper Functions ---------------------------------------------- */

/**
 * Print finite field tests
 */
void print_field_tests(var *result) {
    // Explicit synchronization barrier
    hipDeviceSynchronize();

    // Print results for each limb
    printf("result[0] is: %zu\n", result[0]);
    printf("result[1] is: %zu\n", result[1]);
    printf("result[2] is: %zu\n", result[2]);
    printf("result[3] is: %zu\n\n", result[3]);
}

/* -------------------------- Executing Initialization and Workload Kernels ---------------------------------------------- */

// Execute sum reduction kernel
void execute_sum_reduction(var *a, var *b, var *c, var *d, var *result, var *res_x, var *res_y, var *res_z) {    
    size_t bytes = POINTS * sizeof(int);

    // Allocate dynamic memory
    int *h_v, *h_v_r, *d_v, *d_v_r;
    h_v = (int *)malloc(POINTS * sizeof(int));
    h_v_r = (int *)malloc(POINTS * sizeof(int));
    hipMalloc(&d_v, POINTS * sizeof(int));
    hipMalloc(&d_v_r, POINTS * sizeof(int));

    // Populate array
    for (int i = 0; i < POINTS; i++) {
        h_v[i] = 1;
    }

    // Copy array to device
    hipMemcpy(d_v, h_v, POINTS * sizeof(int), hipMemcpyHostToDevice);

    // Grid size
    int GRID_SIZE = (int)ceil(POINTS / 256);

    // Launch kernels
    // sum_reduction_1<<<GRID_SIZE, 256>>>(d_v, d_v_r);
    // sum_reduction_1<<<1, GRID_SIZE>>>(d_v_r, d_v_r);

    // sum_reduction_2<<<GRID_SIZE, 256>>>(d_v, d_v_r);
    // sum_reduction_2<<<1, GRID_SIZE>>>(d_v_r, d_v_r);

    // sum_reduction_3<<<GRID_SIZE, 256>>>(d_v, d_v_r);
    // sum_reduction_3<<<1, GRID_SIZE>>>(d_v_r, d_v_r);

    // sum_reduction_4<<<GRID_SIZE / 2, 256>>>(d_v, d_v_r);
    // sum_reduction_4<<<1, GRID_SIZE / 2>>>(d_v_r, d_v_r);

    sum_reduction_5<<<GRID_SIZE / 2, 256>>>(d_v, d_v_r);
    sum_reduction_5<<<1, GRID_SIZE / 2>>>(d_v_r, d_v_r);

    // Copy results to host
    hipMemcpy(h_v_r, d_v_r, POINTS * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Accumulated result is: " << h_v_r[0] << endl;
}

/* -------------------------- Main Entry Function ---------------------------------------------- */

int main(int, char**) {
    // Start timer
    auto start = high_resolution_clock::now();

    // Define pointers to uint64_t type
    var *a, *b, *c, *d, *result, *res_x, *res_y, *res_z;

    // Allocate unified memory accessible by host and device
    hipMallocManaged(&a, LIMBS * sizeof(var));
    hipMallocManaged(&b, LIMBS * sizeof(var));
    hipMallocManaged(&c, LIMBS * sizeof(var));
    hipMallocManaged(&d, LIMBS * sizeof(var));
    hipMallocManaged(&result, LIMBS * sizeof(var));
    hipMallocManaged(&res_x, LIMBS * sizeof(var));
    hipMallocManaged(&res_y, LIMBS * sizeof(var));
    hipMallocManaged(&res_z, LIMBS * sizeof(var));

    // Execute kernel functions
    execute_sum_reduction(a, b, c, d, result, res_x, res_y, res_z);

    // Successfull execution of unit tests
    cout << "******* All 'MSM' unit tests passed! **********" << endl;

    // End timer
    auto stop = high_resolution_clock::now();

    // Calculate duraion of execution time 
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "Time taken by function: " << duration.count() << " microseconds\n" << endl; 

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(result);

    cout << "Completed sucessfully!" << endl;

    return 0;
}

/** Developer Doscs 
 * https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
 * https://enccs.github.io/CUDA/3.01_ParallelReduction/#unroll-the-last-warp
*/